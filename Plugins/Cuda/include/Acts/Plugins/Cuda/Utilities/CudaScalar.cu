// This file is part of the Acts project.
//
// Copyright (C) 2020 CERN for the benefit of the Acts project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

#pragma once

#include "Acts/Plugins/Cuda/Utilities/CpuScalar.hpp"

#include <iostream>
#include <memory>

#include "CudaUtils.cu"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cxxabi.h>
#include <thread>


namespace Acts {

template <typename var_t>
class CpuScalar;

template <typename var_t>
class CudaScalar {
 public:
  CudaScalar() {
    ACTS_CUDA_ERROR_CHECK(hipMalloc((var_t**)&m_devPtr, sizeof(var_t)));
  }

  CudaScalar(hipStream_t *s, int dev=0): m_stream(s),m_devID(dev) {
    ACTS_CUDA_ERROR_CHECK(hipMallocAsync((var_t**)&m_devPtr, sizeof(var_t), *m_stream));
  }

  
  CudaScalar(var_t* scalar) {
    ACTS_CUDA_ERROR_CHECK(hipMalloc((var_t**)&m_devPtr, sizeof(var_t)));
    ACTS_CUDA_ERROR_CHECK(
        hipMemcpy(m_devPtr, scalar, sizeof(var_t), hipMemcpyHostToDevice));
  }

  CudaScalar(var_t* scalar, hipStream_t *s, int dev=0):m_stream(s),m_devID(dev) {
    ACTS_CUDA_ERROR_CHECK(hipMallocAsync((var_t**)&m_devPtr, sizeof(var_t),*m_stream));
    ACTS_CUDA_ERROR_CHECK(
                          hipMemcpyAsync(m_devPtr, scalar, sizeof(var_t),
                                          hipMemcpyHostToDevice,*m_stream));
  }

  CudaScalar(const var_t* scalar) {
    ACTS_CUDA_ERROR_CHECK(hipMalloc((var_t**)&m_devPtr, sizeof(var_t)));
    if (m_devPtr == nullptr) {
      throw std::bad_alloc();
    }
    ACTS_CUDA_ERROR_CHECK(
        hipMemcpy(m_devPtr, scalar, sizeof(var_t), hipMemcpyHostToDevice));
  }

  CudaScalar(const var_t* scalar, hipStream_t* s, int dev=0):m_stream(s),m_devID(dev) {
    ACTS_CUDA_ERROR_CHECK(hipMallocAsync((var_t**)&m_devPtr, sizeof(var_t),*m_stream));
    if (m_devPtr == nullptr) {
      throw std::bad_alloc();
    }
    ACTS_CUDA_ERROR_CHECK(
                          hipMemcpyAsync(m_devPtr, scalar, sizeof(var_t),
                                          hipMemcpyHostToDevice, *m_stream));
  }

  ~CudaScalar() {
    if (m_stream) {
      ACTS_CUDA_ERROR_CHECK(hipFreeAsync(m_devPtr,*m_stream));
    } else {
      ACTS_CUDA_ERROR_CHECK(hipFree(m_devPtr));
    }
  }

  var_t* get() { return m_devPtr; }

  void zeros() { hipMemset(m_devPtr, 0, sizeof(var_t)); }

 private:
  var_t* m_devPtr{nullptr};
  hipStream_t* m_stream{nullptr};
  int m_devID {0};
};
}  // namespace Acts
